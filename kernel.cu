﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <csignal>
#include <cstdlib>

hipError_t runGpuMem(unsigned long long memSize);

__global__ void emptyKernel() {
	// do nothing
}


/// <summary>
/// allocates a block of CUDA VRAM and holes on to it, till ctrl +c is pressed. or if 0 is passed display current allocation
/// </summary>
/// <param name="argc">number of arguments</param>
/// <param name="argv">size to allocate in MB or 0 for stats</param>
/// <returns>0 on pass, 1 on fail</returns>
int main(int argc, char* argv[]) {
	unsigned long long memSize = 0;
	hipError_t cudaStatus;

	// get the amount of memory to allocate in MB, default to 256
	if (argc < 2 || sscanf(argv[1], "%llu", &memSize) != 1) {
		
		std::cout << "No size in MB passed in, defaulting to 1024MB\n\n";

		memSize = 1024;

	}

	memSize *= (1024ULL * 1024ULL);  // convert MB to bytes

	// if memSize is 0, just show available VRAM and exit
	if (memSize == 0) {
		size_t freeMem, totalMem;
		hipMemGetInfo(&freeMem, &totalMem);
		std::cout << "Available CUDA VRAM: " << freeMem << " bytes free, " << totalMem << " bytes total.\n";
		return 0;
	}

	// allocate memory and keep an active kernel
	cudaStatus = runGpuMem(memSize);
	if (cudaStatus != hipSuccess) {
		std::cerr << "RunGpuMem failed!\n";
		return 1;
	}

	return 0;
}

/// <summary>
/// allocate a block of VRAM in an empty kernel and wait for CTRL+C
/// </summary>
/// <param name="memSize">amount of memory in MB to allocate</param>
/// <returns>cudaStatus</returns>
hipError_t runGpuMem(unsigned long long memSize) {
	size_t freeMem, totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	std::cout << "Before allocation: " << freeMem << " bytes free, " << totalMem << " bytes total.\n";

	void* gpuMem = nullptr;
	hipError_t cudaStatus = hipMalloc(&gpuMem, memSize);

	if (cudaStatus != hipSuccess) {
		std::cerr << "Error, could not allocate " << memSize << " bytes.\n";
		
	}
	else {
		std::cout << "Allocated " << memSize << " bytes, (" << memSize / (1024ULL * 1024ULL) << " MB).\n";


		hipMemGetInfo(&freeMem, &totalMem);
		std::cout << "After allocation: " << freeMem << " bytes free, " << totalMem << " bytes total.\n";

		// inform user how to exit
		std::cout << "Press CTRL+C to exit...\n";

		while (true) {

			// launch a kernel on the GPU with one thread
			emptyKernel <<<1, 1 >>> ();

			// check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << "\n";
				break;
			}

			// cudaDeviceSynchronize waits for the kernel to finish
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				std::cerr << "CudaDeviceSynchronize returned error code " << cudaStatus << " after launching kernel!\n";
				break;
			}
		}
	}

	// free GPU memory and exit
	hipFree(gpuMem);

	return cudaStatus;
}
